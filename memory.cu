#include "memory.h"


int createcontribAspace(Data* data)
{
	int fullsize;
	int dataSize;
	fullsize =  data->mapInfo.width * data->mapInfo.height;
	dataSize = fullsize * sizeof(int);
	data->contribA = (int *) malloc(dataSize);
	fprintf(data->outlog,"Host memory allocation for contribA  \n");
	return 0;
}


int clearcontribAspace(Data* data)
{
	free(data->contribA);
	//free(data->watershed_id); // need to clear this?

	return 0;
}

int createfilenamespace(Data* data)
{
	    data->heightfile = (char*) malloc(sizeof(char) *100);
	    data->diff_file= (char*) malloc(sizeof(char) *100);
	    data->FDfile = (char*) malloc(sizeof(char) *100);
	    data->FAfile = (char*) malloc(sizeof(char) *100);

	    data->Precipfile = (char*) malloc(sizeof(char) *100);
	    data->Tempfile = (char*) malloc(sizeof(char) *100);

	    data->erofile = (char*) malloc(sizeof(char) *100);
	    data-> incifile = (char*) malloc(sizeof(char) *100);
	    data->gelifile = (char*) malloc(sizeof(char) *100);
	    data->depofile = (char*) malloc(sizeof(char) *100);
	    data->slopefile = (char*) malloc(sizeof(char) *100);

	    data->finesfile = (char*) malloc(sizeof(char) *100);
	    data->stonesfile = (char*) malloc(sizeof(char) *100);
	    data->totbiofile = (char*) malloc(sizeof(char) *100);
	    data->soilTfile = (char*) malloc(sizeof(char) *100);
	    data->nutfile = (char*) malloc(sizeof(char) *100);
	    data->soilMfile = (char*) malloc(sizeof(char) *100);
	    data->soilBfile = (char*) malloc(sizeof(char) *100);

	    data->wCfile = (char*) malloc(sizeof(char) *100);
	    data->wPfile = (char*) malloc(sizeof(char) *100);

	    data->catchmap = (char*) malloc(sizeof(char) *100);
	    data->catchmask = (char*) malloc(sizeof(char) *100);
	    data->contrib = (char*) malloc(sizeof(char) *100);
	    data->rivermaskfile = (char*) malloc(sizeof(char) *100);

	    data->flatfile = (char*) malloc(sizeof(char) *100);

	    data-> logfile = (char*) malloc(sizeof(char) *100);
	    data->outfilename = (char*) malloc(sizeof(char) *100);
	    data->matrixDIR = (char*) malloc(sizeof(char) *100);
	    data->modelcode = (char*) malloc(sizeof(char) *100);
	    data->outputfilefile = (char*) malloc(sizeof(char) *100);

	    data->bedrockfile = (char*) malloc(sizeof(char) *100);

	    data->demfile = (char*) malloc(sizeof(char) *100);
	    data->clim_file = (char*) malloc(sizeof(char) *100);
	    data->dummystring = (char*) malloc(sizeof(char) *100);

	    data->Burnfile = (char*) malloc(sizeof(char) *100);



	    return(1);

}
int createProcessMatrices(Data* data)
{
  int fullsize;
  int dataSize;
  int dataSizeInt;

  fullsize =  data->mapInfo.width * data->mapInfo.height;
  dataSize = fullsize * sizeof(double);
  dataSizeInt = fullsize * sizeof(int);

// these are the static grids in which data is stored from one iteration to the next ie. these are ONLY freed at the end of the simulation

  checkCudaErrors(hipHostMalloc((void **)&data->prevdem, dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->diffdem, dataSize));

  checkCudaErrors(hipHostMalloc((void **)&data->fd, dataSizeInt ));
  checkCudaErrors(hipHostMalloc((void **)&data->SFD, dataSizeInt ));
  fprintf(data->outlog, "Flow direction (fd and SFD) space on host allocated %s\n", hipGetErrorString(hipGetLastError()));

  checkCudaErrors(hipHostMalloc((void **)&data->fa,           dataSize));
  fprintf(data->outlog, "Flow accumulation space on host allocated %s\n", hipGetErrorString(hipGetLastError()));

  checkCudaErrors(hipHostMalloc((void **)&data->SlopePtr,     dataSize));

  checkCudaErrors(hipHostMalloc((void **)&data->flatmask, dataSizeInt  ));

// room to store the slopes and proportions in all directions
  checkCudaErrors(hipHostMalloc((void **)&data->Slopes,  dataSize * 8 ));
  checkCudaErrors(hipHostMalloc((void **)&data->prop,  dataSize * 8  ));


  checkCudaErrors(hipHostMalloc((void **)&data->runoffweight, dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->stonePtr,     dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->finesPtr,     dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->soilMPtr,     dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->soilBPtr,     dataSize));
  //checkCudaErrors(hipHostMalloc((void **)&data->soilTPtr,     dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->nutPtr,       dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->TotBPtr,      dataSize));

  checkCudaErrors(hipHostMalloc((void **)&data->eroPtr,       dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->geliPtr,      dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->inciPtr,      dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->depoPtr,      dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->dz,           dataSize));

  checkCudaErrors(hipHostMalloc((void **)&data->weatherC,     dataSize));
  checkCudaErrors(hipHostMalloc((void **)&data->weatherP,     dataSize));

  fprintf(data->outlog, "All hosts matrices memory allocated %s\n", hipGetErrorString(hipGetLastError()));

  return 0;

}


int deleteProcessMatrices(Data* data)
{

	  //checkCudaErrors(hipHostFree(data->dem));
	  checkCudaErrors(hipHostFree(data->prevdem));
	  checkCudaErrors(hipHostFree(data->diffdem));
	  checkCudaErrors(hipHostFree(data->dz));
	  checkCudaErrors(hipHostFree(data->fd));
	  checkCudaErrors(hipHostFree(data->fa));
	  checkCudaErrors(hipHostFree(data->SlopePtr));

	  checkCudaErrors(hipHostFree(data->rainmat));
	  checkCudaErrors(hipHostFree(data->tempmat));
	  checkCudaErrors(hipHostFree(data->last_rainmat));
	  checkCudaErrors(hipHostFree(data->last_tempmat));

	  checkCudaErrors(hipHostFree(data->mask));
	  checkCudaErrors(hipHostFree(data->flatmask));
	  //checkCudaErrors(hipHostFree(data->Flow_C));
	  // free the slope and prop matrices
	  checkCudaErrors(hipHostFree(data->Slopes));
	  checkCudaErrors(hipHostFree(data->prop));

	  checkCudaErrors(hipHostFree(data->runoffweight));
	  checkCudaErrors(hipHostFree(data->stonePtr));
	  checkCudaErrors(hipHostFree(data->finesPtr));
	  checkCudaErrors(hipHostFree(data->soilMPtr));
	  checkCudaErrors(hipHostFree(data->soilBPtr));
	  checkCudaErrors(hipHostFree(data->soilTPtr));
	  checkCudaErrors(hipHostFree(data->nutPtr));
	  checkCudaErrors(hipHostFree(data->TotBPtr));

	  checkCudaErrors(hipHostFree(data->eroPtr)); 
	  checkCudaErrors(hipHostFree(data->geliPtr));
	  checkCudaErrors(hipHostFree(data->inciPtr));
	  checkCudaErrors(hipHostFree(data->depoPtr));
	  // checkCudaErrors(hipHostFree(data->dz));
	  checkCudaErrors(hipHostFree(data->weatherC));
	  checkCudaErrors(hipHostFree(data->weatherP));

	  fprintf(data->outlog, "All hosts matrices memory freed \n");

	return 0;
}


//////////////////////////////////////////////////////////////////////////////
// Setup store for catchment data ( needed for summary outputs etc)
//////////////////////////////////////////////////////////////////////////////

int createCatchmentSpace(Data* data, Catchment* Catchments) {
	//allocate space for catchment data and selective list and set values to zero
	Catchments->watershed_id = (int *) calloc(sizeof(int) , data->mapInfo.height * data->mapInfo.width);
	Catchments->mask = (int *) calloc(sizeof(int),  data->mapInfo.height * data->mapInfo.width); // all mask values set to zero

	fprintf(data->outlog, "Catchment space allocated \n");
	return 0;
}


void createSoilTfromformula(Data* data){

	int cell;

	checkCudaErrors(hipHostMalloc((void **)&data->soilTPtr, data->mapInfo.width * data->mapInfo.height * sizeof(double)));

	  // what if the lowest cell is currently in the grid!
	  for (int i = 0; i < data->mapInfo.height; i++){
			for (int j = 0; j < data->mapInfo.width; j++) {
				cell = 	i*data->mapInfo.width + j;
					if ( (data->dem[cell]) > 0 ) { data->soilTPtr[cell] = ( (data->dem[cell] - 400) / 1400 ) * 5;}
						else { data->soilTPtr[cell] = 0.0;}

				  }
	  }
	  printf( "Soil Thickness Data Created \n");
	  return;
}


int createmask(Data* data)
{

int width = data->mapInfo.width;
int height = data->mapInfo.height ;
int fullsize = width * height;
double nodataold;

nodataold = data->mapInfo.nodata;
printf("DEM old no data value = %.6f will be reset to -9999\n", data->mapInfo.nodata);

checkCudaErrors(hipHostMalloc((void **)&data->mask,     fullsize*sizeof(int)  ));
	for (int x = 0; x < fullsize; x++) {
		data->mask[x] = 1;

		if (data->dem[x] == -9999)
		{
			data->mask[x] = 0;
			data->dem[x] = -9999; //reset the no data value to -9999
		}

	}
	data->mapInfo.nodata = -9999;

 return 1;
}
