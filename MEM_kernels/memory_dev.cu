
#include "memory_dev.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime_api.h"

//#include "helper_timer.h"
//#include "helper_functions.h"
//#include "hip/hip_runtime_api.h"
//#include "hip/hip_runtime.h"
//#include <>



void setslopesspace(Data* data, Data* device)
{
	 size_t freenow, total;
	 int fullsize;
	 int ncell_x = data->mapInfo.width;
	 int ncell_y = data->mapInfo.height;
	 fullsize= ncell_x * ncell_y * 9;

	 //checkCudaErrors( hipMalloc((void**)&(device->Slopes), fullsize * sizeof(double)) );


	  hipMemGetInfo(&freenow, &total);
	  fprintf(data->outlog, "CUDA card free after Slopes space allocated: %zd total: %zd \n",freenow/1024,total/1024);
	  fprintf(data->outlog, "slopes space allocated:on host and device  %s\n", hipGetErrorString(hipGetLastError()));
}

void setdevicespace_FD(Data* data, Data* device)
{
	 size_t freenow, total;
	 int fullsize;
	 int doublefull;
	 int ncell_x = data->mapInfo.width;
	 int ncell_y = data->mapInfo.height;
	 fullsize= ncell_x * ncell_y;
	 doublefull = fullsize* sizeof(double) *8;

	 checkCudaErrors( hipMalloc((void**)&(device->fd), fullsize * sizeof(int)) );
	 checkCudaErrors( hipMalloc((void**)&(device->SFD), fullsize * sizeof(int)) );

	 checkCudaErrors( hipMalloc((void**)&(device->dx), 9 * sizeof(int)) );
	 checkCudaErrors( hipMalloc((void**)&(device->dy), 9 * sizeof(int)) );
	 checkCudaErrors( hipMalloc((void**)&(device->shortest_paths), fullsize * sizeof(float))  );
	 checkCudaErrors( hipMalloc((void**)&(device->lowHeight),      fullsize * sizeof(double)) );
	 checkCudaErrors( hipMalloc((void**)&(device->watershed_id),     fullsize * sizeof(int))    );
	 checkCudaErrors( hipMalloc((void**)&(device->flatmask),     fullsize * sizeof(int))    );
	 fprintf(data->outlog, "FD: setdevicespace0:%s\n", hipGetErrorString(hipGetLastError()));

	 checkCudaErrors( hipMalloc((void**)&(device->Slopes), doublefull) );
	 checkCudaErrors( hipMalloc((void**)&(device->prop),   doublefull) );
	 fprintf(data->outlog, "FD: setdevicespace1:%s\n", hipGetErrorString(hipGetLastError()));

	  hipMemGetInfo(&freenow, &total);
	  fprintf(data->outlog, "Memory on CUDA card free after FD space allocated: %zd total: %zd \n",freenow/1024,total/1024);
	  fprintf(data->outlog, "FD: setdevicespace2:%s\n", hipGetErrorString(hipGetLastError()));

}


void cleardevicespace_FD(Data* data, Data* device)
{
	size_t freenow, total;

		hipFree(device->fd);
		hipFree(device->SFD);
		// hipFree(device->Slopes); //keep on device
		// hipFree(device->prop); //keep on device

		hipFree(device->dx);
		hipFree(device->dy);
		hipFree(device->shortest_paths);
		hipFree(device->lowHeight);
		hipFree(device->watershed_id);
		hipFree(device->flatmask);
		fprintf(data->outlog, "FD: error after FD clear :%s\n", hipGetErrorString(hipGetLastError()));

		hipMemGetInfo(&freenow, &total);
		fprintf(data->outlog, "FD: Memory on CUDA card free after FD space freed: %zd total: %zd \n\n",freenow/1024,total/1024);

}

void setdevicespace_FA(Data* data, Data* device)
{
	int full_size;
	int ncell_x = data->mapInfo.width;
	int ncell_y = data->mapInfo.height;
	full_size= ncell_x * ncell_y;

	checkCudaErrors( hipMalloc( (void**) &device->runoffweight, full_size * sizeof(double)) );

	checkCudaErrors( hipMalloc( (void**) &device->rainmat, full_size * sizeof(double)) );
	checkCudaErrors( hipMalloc( (void**) &device->tempmat, full_size * sizeof(double)) );

	checkCudaErrors( hipMalloc( (void**) &device->fa, full_size * sizeof(double)) );
	checkCudaErrors( hipMalloc( (void**) &device->fd, full_size * sizeof(int)) );

	//checkCudaErrors( hipMalloc( (void**) &device->Slopes, full_size * 8 * sizeof(double)) );

	checkCudaErrors( hipMalloc( (void**) &device->stonePtr, full_size * sizeof(double)) );
	checkCudaErrors( hipMalloc( (void**) &device->TotBPtr, full_size * sizeof(double)) );
	checkCudaErrors( hipMalloc( (void**) &device->soilMPtr, full_size * sizeof(double) ));

	// now copy the necessary data - these will not overlap becasue they are all on the same stream

	//checkCudaErrors(hipSetDevice(0));
	checkCudaErrors( hipMemcpy( device->dem, data->dem, full_size * sizeof(double), hipMemcpyHostToDevice)) ; // copy the non-raised DEM back to GPU
	checkCudaErrors( hipMemcpy( device->fd, data->fd, full_size * sizeof(int), hipMemcpyHostToDevice)) ;
	checkCudaErrors( hipMemcpy( device->runoffweight, data->runoffweight, full_size * sizeof(double), hipMemcpyHostToDevice)) ;

	checkCudaErrors( hipMemcpy( device->rainmat, data->rainmat, full_size * sizeof(double), hipMemcpyHostToDevice)) ;
	checkCudaErrors( hipMemcpy( device->tempmat, data->tempmat, full_size * sizeof(double), hipMemcpyHostToDevice)) ;

	checkCudaErrors( hipMemcpy( device->stonePtr, data->stonePtr, full_size * sizeof(double), hipMemcpyHostToDevice)) ;
	checkCudaErrors( hipMemcpy( device->TotBPtr, data->TotBPtr, full_size * sizeof(double), hipMemcpyHostToDevice)) ;
	checkCudaErrors( hipMemcpy( device->soilMPtr, data->soilMPtr, full_size * sizeof(double), hipMemcpyHostToDevice)) ;

#ifndef PRODUCTION_RUN
	printf("FA: setdevicespace_FA:%s\n", hipGetErrorString(hipGetLastError()));
#endif

	fprintf(data->outlog, "FA: setdevicespace_FA:%s\n", hipGetErrorString(hipGetLastError()));
	fflush(data->outlog);
}

void cleardevicespace_FA(Data* data, Data* device)
{
	int full_size;
	int ncell_x = data->mapInfo.width;
	int ncell_y = data->mapInfo.height;
	full_size = ncell_x * ncell_y;

	size_t freenow, total;
	checkCudaErrors(hipMemcpy(data->prevfd, device->fd, full_size * sizeof(int), hipMemcpyDeviceToHost));
	hipFree(device->fd);
	hipFree(device->runoffweight);
	hipFree(device->fa);

	//hipFree(device->contribA); // free it here as it is no longer needed)

	hipMemGetInfo(&freenow, &total);
	fprintf(data->outlog, "FA: Memory on CUDA card free after FA space freed: %zd total: %zd \n\n",freenow/1024,total/1024);

	fprintf(data->outlog, "FA: cleardevicespace_FA:%s\n", hipGetErrorString(hipGetLastError()));
}


void setdevicespace_Process(Data* data, Data* device)
{
	size_t freenow, total;
	int full_size;
	int ncell_x = data->mapInfo.width;
	int ncell_y = data->mapInfo.height;
	full_size= ncell_x * ncell_y;



	checkCudaErrors( hipMalloc( (void**) &device->fa,       full_size * sizeof(double)) );
	checkCudaErrors( hipMalloc( (void**) &device->fd,       full_size * sizeof(int))    );
	checkCudaErrors( hipMalloc( (void**) &device->SFD,      full_size * sizeof(int))    );

	checkCudaErrors( hipMalloc( (void**) &device->dz,       full_size * sizeof(double)) ); // create room for product dz
	checkCudaErrors( hipMalloc( (void**) &device->finesPtr, full_size * sizeof(double)) );
	checkCudaErrors( hipMalloc( (void**) &device->soilTPtr, full_size * sizeof(double)) );
	checkCudaErrors( hipMalloc( (void**) &device->nutPtr,   full_size * sizeof(double)) );
	checkCudaErrors( hipMalloc( (void**) &device->soilBPtr, full_size * sizeof(double)) );
	checkCudaErrors( hipMalloc( (void**) &device->eroPtr,   full_size * sizeof(double)) );
	checkCudaErrors( hipMalloc( (void**) &device->geliPtr,  full_size * sizeof(double)) );
	checkCudaErrors( hipMalloc( (void**) &device->inciPtr,  full_size * sizeof(double)) );
	checkCudaErrors( hipMalloc( (void**) &device->depoPtr,  full_size * sizeof(double)) );
	checkCudaErrors( hipMalloc( (void**) &device->weatherC, full_size * sizeof(double)) );
	checkCudaErrors( hipMalloc( (void**) &device->weatherP, full_size * sizeof(double)) );

		fprintf(data->outlog, "MOD: setdevicespace_Process :%s\n", hipGetErrorString(hipGetLastError()));

		// stones, TotBio, soilM plus dem, slopes and mask still on device
		checkCudaErrors( hipMemcpy ( device->fa,       data->fa,         full_size * sizeof(double), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->fd,       data->fd,         full_size * sizeof(int),    hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->SFD,      data->fd,         full_size * sizeof(int),    hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->SlopePtr,  data->SlopePtr,  full_size * sizeof(double), hipMemcpyHostToDevice) );

		checkCudaErrors( hipMemcpy ( device->finesPtr, data->finesPtr,   full_size * sizeof(double), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->soilTPtr, data->soilTPtr,   full_size * sizeof(double), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->nutPtr,   data->nutPtr,     full_size * sizeof(double), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->soilBPtr, data->soilBPtr,   full_size * sizeof(double), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->eroPtr,   data->eroPtr,     full_size * sizeof(double), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->geliPtr,  data->geliPtr,    full_size * sizeof(double), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->inciPtr,  data->inciPtr,    full_size * sizeof(double), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->depoPtr,  data->depoPtr,    full_size * sizeof(double), hipMemcpyHostToDevice) );

		checkCudaErrors( hipMemcpy ( device->weatherC,  data->weatherC,    full_size * sizeof(double), hipMemcpyHostToDevice) );
		checkCudaErrors( hipMemcpy ( device->weatherP,  data->weatherP,    full_size * sizeof(double), hipMemcpyHostToDevice) );

		fprintf(data->outlog, "MOD: Matrix memcopy operations :%s\n", hipGetErrorString(hipGetLastError()));

		hipMemGetInfo(&freenow, &total);
		fprintf(data->outlog, "MOD: Memory on CUDA card free after model matrix space allocated: %zd total: %zd \n",freenow/1024,total/1024);
}

void cleardevicespace_Process(Data* data, Data* device)
{
	size_t freenow, total;

	hipFree(device->fa);
	hipFree(device->fd);
	hipFree(device->SFD);
	hipFree(device->fdmod);

	hipFree(device->Slopes);
	//hipFree(device->SlopePtr); // do not free here as it will not be redeclared.
	hipFree(device->prop);

	hipFree(device->rainmat);
	hipFree(device->tempmat);

	hipFree(device->dz);

	hipFree(device->finesPtr);
	hipFree(device->soilTPtr);
	hipFree(device->nutPtr);
	hipFree(device->soilBPtr);
	hipFree(device->eroPtr);
	hipFree(device->geliPtr);
	hipFree(device->inciPtr);
	hipFree(device->depoPtr);
	hipFree(device->weatherC);
	hipFree(device->weatherP);

	// free after being left at end of FA routines.
	hipFree(device->stonePtr);
	hipFree(device->TotBPtr);
	hipFree(device->soilMPtr);

	hipMemGetInfo(&freenow, &total);
	fprintf(data->outlog, "MOD: Memory on CUDA card free after model space freed: %zd total: %zd \n",freenow/1024,total/1024);
	fprintf(data->outlog, "MOD: Clear matrix operations :%s\n", hipGetErrorString(hipGetLastError()));

}

int copyMask(Data* data, Data* device)
{


	 int fullsize;
	 int ncell_x = data->mapInfo.width;
	 int ncell_y = data->mapInfo.height;
	 fullsize= ncell_x * ncell_y;

	 checkCudaErrors( hipMalloc( (void**) &device->mask, fullsize * sizeof(int)) ); // create space for the mask

	 checkCudaErrors( hipMemcpy(device->mask, data->mask, fullsize * sizeof(int), hipMemcpyHostToDevice) );  // copy back flag
	 fprintf(data->outlog, "Mask data sent to device %s\n", hipGetErrorString(hipGetLastError()));

	thrust::device_ptr<int> activecells = thrust::device_pointer_cast(device->mask);
	data->activecells  = thrust::count(activecells, activecells + fullsize, 1);

#ifndef PRODUCTION_RUN
	printf("No of active cells = %d \n", data->activecells);
#endif

	return 0;
}

int copylastclimate(Data* data, Data* device)
{

	 int fullsize;
	 int ncell_x = data->mapInfo.width;
	 int ncell_y = data->mapInfo.height;
	 fullsize= ncell_x * ncell_y;

		checkCudaErrors( hipMalloc( (void**) &device->last_rainmat, fullsize * sizeof(double)) );
		checkCudaErrors( hipMalloc( (void**) &device->last_tempmat, fullsize * sizeof(double)) );

		checkCudaErrors( hipMemcpy( device->last_rainmat, data->last_rainmat, fullsize * sizeof(double), hipMemcpyHostToDevice)) ;
		checkCudaErrors( hipMemcpy( device->last_tempmat, data->last_tempmat, fullsize * sizeof(double), hipMemcpyHostToDevice)) ;

		fprintf(data->outlog, "Last climate data sent to device %s\n", hipGetErrorString(hipGetLastError()));

		//initial host matrices no longer needed. Will direct copy on only on GPU subsequently
		hipFree(data->last_rainmat);
		hipFree(data->last_tempmat);

		fprintf(data->outlog, "Error here? %s\n", hipGetErrorString(hipGetLastError()));

	return 0;
}



void createDeviceSpace(Data* data, Data* device)
{
	size_t freenow, total;

	 int fullsize;
	 int ncell_x = data->mapInfo.width;
	 int ncell_y = data->mapInfo.height;
	 fullsize= ncell_x * ncell_y;

	 //hipDeviceReset();

	  checkCudaErrors( hipMalloc((void **)&(device->dem), fullsize * sizeof(double)) );
	  checkCudaErrors( hipMalloc((void **)&(device->SlopePtr), fullsize * sizeof(double)) );
	  checkCudaErrors( hipMalloc((void **)&(device->summary), fullsize * sizeof(double)) );

	fprintf(data->outlog,"Allocated DEM and slope matrices on device :%s\n", hipGetErrorString(hipGetLastError()));

	hipMemGetInfo(&freenow, &total);
	fprintf(data->outlog,"Memory on CUDA card free after device DEM and slope grids allocated: %zd total: %zd \n",freenow/1024,total/1024);

#ifndef PRODUCTION_RUN
	printf("Device space created \n");
#endif

}


int clearDeviceSpace(Data* data, Data* device)
{
	size_t freenow, total;

		//hipFree(device->dem);
		hipFree(device->SlopePtr);
		hipFree(device->summary);
		hipFree(device->Slopes);


	hipMemGetInfo(&freenow, &total);
#ifndef PRODUCTION_RUN
	printf("Memory on CUDA card free after DEM and slope device grids space freed: %d total: %d \n",freenow/1024,total/1024);
#endif

	fprintf(data->outlog,"Memory on CUDA card free after DEM and slope device grids space freed: %zd total: %zd \n",freenow/1024,total/1024);





	return 0;
}

int zerogrids(Data* data)
{

	memset(data->eroPtr, 0.0, sizeof(data->eroPtr));
	memset(data->geliPtr, 0.0, sizeof(data->eroPtr));
	memset(data->inciPtr, 0.0, sizeof(data->inciPtr));
	memset(data->depoPtr, 0.0, sizeof(data->depoPtr));

	memset(data->fa, 0.0, sizeof(data->fa));

	return 0;
}

int copytolastclimate(Data* data, Data* device)
{
	int fullsize;
	int ncell_x = data->mapInfo.width;
	int ncell_y = data->mapInfo.height;
	fullsize= ncell_x * ncell_y;

	checkCudaErrors( hipMemcpy( device->last_rainmat, device->rainmat, fullsize * sizeof(double), hipMemcpyDeviceToDevice)) ;
	checkCudaErrors( hipMemcpy( device->last_tempmat, device->tempmat, fullsize * sizeof(double), hipMemcpyDeviceToDevice)) ;

	return 0;
}
