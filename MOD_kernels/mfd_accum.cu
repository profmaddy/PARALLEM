#include "hip/hip_runtime.h"
#include "mfd_accum.h"
//#include "Directions.h"
//#include "Data.h"
//#include "config.h"
//#include "lem.h"

__device__ int inghbr(int client, int nghbr, int gridCols)
{
    switch (nghbr)
    {
        case EAST:
            return client + 1;
        case SOUTHEAST:
            return client + gridCols + 1;
        case SOUTH:
            return client + gridCols;
        case SOUTHWEST:
            return client + gridCols - 1;
        case WEST:
            return client - 1;
        case NORTHWEST:
            return client - gridCols - 1;
        case NORTH:
            return client - gridCols;
        case NORTHEAST:
            return client - gridCols + 1;
        default:
            return -1;
    }
}

__global__ void MFDsedbud(int *mask, double *hv, int *fd, double* props, double *diffuse, double *conc, double *geli,  double *depo, int *ok,
						  unsigned int *progressd2,  int* localOK, double ddk, double dck, double dgk,
						  double * finesPtr, double *stonePtr, double *nutPtr, double *soilTPtr , int gridCols, int gridRows)
{
	  int irow, icol;
	  irow = blockIdx.y * blockDim.y + threadIdx.y;
	  icol = blockIdx.x * blockDim.x + threadIdx.x;
	  // Are we outside the DEM?
	  if(icol >= gridCols || irow >= gridRows) // if outside of DEM nothing to do
	    return;
      if(icol == 0 || irow == 0 ) return; // if outside of DEM nothing to do

	int self;
    self = irow * gridCols + icol;
	if (mask[self] == 0) return;

    double accum_d, accum_c, accum_g;
	unsigned int theval = 99999999;
    int   nie, nise, nis, nisw, niw, ninw, nin, nine;
    double nce_from, nce_nutfrom;
    double ncse_from, ncse_nutfrom;
    double ncs_from, ncs_nutfrom;
    double ncsw_from, ncsw_nutfrom;
    double ncw_from, ncw_nutfrom;
    double ncnw_from, ncnw_nutfrom;
    double ncn_from, ncn_nutfrom;
    double ncne_from, ncne_nutfrom;
    double  fromEprop,  fromSEprop,  fromSprop,  fromSWprop,  fromWprop , fromNWprop ,  fromNprop ,  fromNEprop;
    int cnfd;
    int proploc;
    
    accum_d = 0.0;
    accum_c = 0.0;
    accum_g = 0.0;
        
    nce_from = nce_nutfrom = 0.0;
    ncse_from = ncse_nutfrom =  0.0;
    ncs_from = ncs_nutfrom = 0.0;
    ncsw_from =ncsw_nutfrom = 0.0;
    ncw_from = ncw_nutfrom = 0.0;
    ncnw_from = ncnw_nutfrom = 0.0;
    ncn_from = ncn_nutfrom = 0.0;
    ncne_from = ncne_nutfrom = 0.0;

    if (ok[self] != 0) return;

	if (localOK[self] == 1) {
      ok[self] = 1;
      return;
    }

    nie  = inghbr(self, EAST,      gridCols);
    nise = inghbr(self, SOUTHEAST, gridCols);
    nis  = inghbr(self, SOUTH,     gridCols);
    nisw = inghbr(self, SOUTHWEST, gridCols);
    niw  = inghbr(self, WEST,      gridCols);
    ninw = inghbr(self, NORTHWEST, gridCols);
    nin  = inghbr(self, NORTH,     gridCols);
    nine = inghbr(self, NORTHEAST, gridCols);

    /* ---- */
    cnfd = fd[nie];  // cnfd is current flow direction of contributing cell to east
    
    if (cnfd & WEST)
    {
        if (!ok[nie]) return;
            /* if (!ok[nie]) return;
            proploc = (nie * 8) + 6;// flowing west
            addhere = fa[nie] * props[proploc];
            accum += addhere;*/
                proploc = (nie * 8) + 6;// flowing west
                accum_d  += props[proploc]  * (diffuse[nie] * (1-ddk));
		        accum_c  += props[proploc] * (conc[nie]    * (1-dck)) ;
		        accum_g  += props[proploc] * (geli[nie]    * (1-dgk));
		        nce_from       = props[proploc] * stonePtr[nie] ;
		        nce_nutfrom    = props[proploc] * nutPtr[nie] ;
    }

    cnfd = fd[nise];
    
    if (cnfd & NORTHWEST)
    {
        if (!ok[nise]) return;
            /*if (!ok[nise]) return;
            proploc = (nise * 8) + 7;// flowing northwest
            addhere = fa[nise] * props[proploc];
            accum += addhere;*/
                proploc = (nise * 8) + 7;// flowing northwest
        	    accum_d  += props[proploc] * (diffuse[nise] * (1-ddk));
        	    accum_c  += props[proploc] * (conc[nise]     * (1-dck)) ;
        	    accum_g  += props[proploc] * (geli[nise]       * (1-dgk));
        	    ncse_from       = props[proploc] * stonePtr[nise] ;
        	    ncse_nutfrom    = props[proploc] * nutPtr[nise] ;
    }

    cnfd = fd[nis];
    if (cnfd & NORTH)
    {
        if (!ok[nis]) return;
        /*if (!ok[nis]) return;
        proploc = (nis * 8) + 0;//flowing north
        addhere = fa[nis] * props[proploc];
        accum += addhere;*/
                proploc = (nis * 8) + 0;//flowing north
        	    accum_d  += props[proploc] * (diffuse[nis] * (1-ddk));
        	    accum_c  += props[proploc] * (conc[nis]    * (1-dck)) ;
        	    accum_g  += props[proploc] * (geli[nis]      * (1-dgk));
        	    ncs_from       = props[proploc] * stonePtr[nis] ;
        	    ncs_nutfrom    = props[proploc] * nutPtr[nis] ;
    }

    cnfd = fd[nisw];
    if (cnfd & NORTHEAST)
    {
        if (!ok[nisw]) return;
        /*if (!ok[nisw]) return;
        proploc = (nisw * 8) + 1;//flowing northeast
        addhere = fa[nisw] * props[proploc];
        accum += addhere;*/
                proploc = (nisw * 8) + 1;//flowing northeast
                accum_d  += props[proploc] *  (diffuse[nisw] * (1-ddk));
                accum_c  += props[proploc] * (conc[nisw]     * (1-dck)) ;
                accum_g  += props[proploc] * (geli[nisw]       * (1-dgk));
	            ncsw_from       = props[proploc] * stonePtr[nisw] ;
	            ncsw_nutfrom    = props[proploc] * nutPtr[nisw] ;
    }

    cnfd = fd[niw];
    if (cnfd & EAST)
    {
        if (!ok[niw]) return;
        /*if (!ok[niw]) return;
        proploc = (niw * 8) + 2; // flowing east
        addhere = fa[niw] * props[proploc];
        accum += addhere;*/
                proploc = (niw * 8) + 2; // flowing east
                accum_d  += props[proploc] * (diffuse[niw] * (1-ddk));
                accum_c  += props[proploc] * (conc[niw]     * (1-dck)) ;
                accum_g  += props[proploc] * (geli[niw]       * (1-dgk));
                ncw_from       = props[proploc] * stonePtr[niw] ;
                ncw_nutfrom    = props[proploc] * nutPtr[niw] ;
    }

    cnfd = fd[ninw];
    if (cnfd & SOUTHEAST)
    {
        if (!ok[ninw]) return;
        /*if (!ok[ninw]) return;
        proploc = (ninw * 8) + 3;//flowing southeast
        addhere = fa[ninw] * props[proploc];
        accum += addhere;*/
                proploc = (ninw * 8) + 3;//flowing southeast
        	    accum_d  += props[proploc] * (diffuse[ninw] * (1-ddk));
        	    accum_c  += props[proploc] * (conc[ninw]     * (1-dck)) ;
        	    accum_g  += props[proploc] * (geli[ninw]       * (1-dgk));
        	    ncnw_from       = props[proploc] * stonePtr[ninw] ;
        	    ncnw_nutfrom    = props[proploc] * nutPtr[ninw] ;
    }

    cnfd = fd[nin];
    if (cnfd & SOUTH)
    {
        if (!ok[nin]) return;
        /*if (!ok[nin]) return;
        proploc = (nin * 8) + 4;//flowing south
        addhere = fa[nin] * props[proploc];
        accum += addhere;*/
                proploc = (nin * 8) + 4;//flowing south
        	    accum_d  += props[proploc] * (diffuse[nin] * (1-ddk));
        	    accum_c  += props[proploc] * (conc[nin]     * (1-dck)) ;
        	    accum_g  += props[proploc] * (geli[nin]       * (1-dgk));
        	    ncn_from       = props[proploc] * stonePtr[nin] ;
        	    ncn_nutfrom    = props[proploc] * nutPtr[nin] ;
    }

    cnfd = fd[nine];
    if (cnfd & SOUTHWEST)
    {
        if (!ok[nine]) return;
       /* if (!ok[nine]) return;
        proploc = (nine * 8) + 5;//flowing southwest
        addhere = fa[nine] * props[proploc];
        accum += addhere;*/
                proploc = (nine * 8) + 5;//flowing southwest
        	    accum_d  += props[proploc] * (diffuse[nine] * (1-ddk));
        	    accum_c  += props[proploc] * (conc[nine]     * (1-dck)) ;
        	    accum_g  += props[proploc] * (geli[nine]       * (1-dgk));
        	    ncne_from       = props[proploc] * stonePtr[nine] ;
        	    ncne_nutfrom    = props[proploc] * nutPtr[nine] ;
    }

    depo[self] =  ((accum_d + diffuse[self])* ddk) + ((accum_c + conc[self]) * dck) + ((accum_g + geli[self]) * dgk);

    // replace original deposit routine to calculate new proportions of fines, stones and nutrients

    double sedthick = (soilTPtr[self]) + (depo[self]) ;
	if (sedthick != 0.0)
	{
		double old = (soilTPtr[self]) / sedthick ;

		double input = 1-old;
		//double input = (depo[self]) /weight;

		double propStonefrom = ( nce_from + ncse_from + ncs_from + ncsw_from + ncw_from + ncnw_from + ncn_from + ncne_from) ;/// ndirections ;
        double propNutfrom = ( nce_nutfrom + ncse_nutfrom + ncs_nutfrom + ncsw_nutfrom + ncw_nutfrom + ncnw_nutfrom + ncn_nutfrom + ncne_nutfrom) ;

        stonePtr[self] =   (old * (stonePtr[self]))   + (input * (propStonefrom)) ;

		if (stonePtr[self] < 0.01) stonePtr[self] = 0.01;
		else if (stonePtr[self] > 99.9) stonePtr[self] = 99.9;

		finesPtr[self] = 100 - stonePtr[self];  // now same as calculation in weathering
		//finesPtr[self] = old * (finesPtr[self]) + input * (finesPtr[from]);

		//nutPtr[self] =   (old * (nutPtr[self]))   + (input * propNutfrom) ;
	}
	double toterosion = diffuse[self]+conc[self]+geli[self];
	if (toterosion >= sedthick) soilTPtr[self]= 0.0;
	if (sedthick > toterosion) soilTPtr[self] = sedthick - toterosion;

    localOK[self] = 1;
    atomicInc(progressd2, theval);
}

int processtheGrid(Data* data, Data* device, int loopMax, int percent, int gridRows, int gridColumns, int* okGrid,  int* localOK, int blockRows, int blockColumns, int dimBlock3, int* doneP,
													double ddk, double dck, double  dgk)

{
  int loopForever = (loopMax < 0) ? 1 : 0;

  int *ok;
  int *localOK_d;


  unsigned int progressh2, *progressd2;
  int gridProgress;

  //allocate GPU memory
    checkCudaErrors(hipMalloc((void **) &ok,                   gridRows * gridColumns * sizeof(int))    );
    checkCudaErrors(hipMalloc((void **) &localOK_d,     gridRows * gridColumns * sizeof(int))   );
    checkCudaErrors(hipMalloc((void **) &progressd2,      sizeof(progressh2)) );
    fprintf(data->outlog, "MOD: sedmfd memory allocation :%s\n", hipGetErrorString(hipGetLastError()));

  gridProgress = 0;
  int loop = 0;

  int grid1 = data->mapInfo.width  / (blockColumns ) + 1;
  int grid2 = data->mapInfo.height / (blockRows ) + 1;

  //printf("blockColumns = %d  blockRows = %d  grid1 = %d  grid2 = %d \n", blockColumns, blockRows, grid1, grid2);

  do {
	  	  dim3 dimGrid(grid1, grid2, 1);
	  	  dim3 dimBlock(blockColumns, blockRows, dimBlock3);
  //printf("Grid is %d by %d by %d\n", dimGrid1, dimGrid2, dimGrid3);
	      int oneZero = 0;
	      int oneZero2 = 0;

	      //copy grids to GPU
	        checkCudaErrors(hipMemcpy(ok,                    okGrid,    gridRows * gridColumns * sizeof(int),    hipMemcpyHostToDevice) );
	        checkCudaErrors(hipMemcpy(localOK_d,      localOK,  gridRows * gridColumns * sizeof(int),    hipMemcpyHostToDevice) );

	        do {
	        		progressh2 = 0;
	        		oneZero2 = oneZero;

	        		checkCudaErrors(hipMemcpy(progressd2, &progressh2, sizeof(progressh2), hipMemcpyHostToDevice) );

//	        		MFDsedbud(double *hv, int *fd, double *diffuse, double *conc, double *geli,  int *ok, unsigned int *progressd,  int* localOK,
// 						double ddk, double dck, double dgk, double * finesPtr, double *stonePtr, double *nutPtr, double *soilTPtr , int gridCols)

	        		MFDsedbud<<<dimGrid, dimBlock>>>(device->mask, device->dem, device->fd, device->prop, device->eroPtr, device->inciPtr, device->geliPtr, device->depoPtr,  ok, progressd2, localOK_d,
	        				                                                    ddk, dck, dgk, device->finesPtr, device->stonePtr, device->nutPtr, device->soilTPtr, gridColumns, gridRows);

	        		//fprintf(data->outlog, "MOD: MFDsedbud rtn :%s\n", hipGetErrorString(hipGetLastError()));
	        		//printf("MOD: MFDsedbud rtn :%s\n", hipGetErrorString(hipGetLastError()));
	        		checkCudaErrors(hipMemcpy(&progressh2, progressd2, sizeof(progressh2), hipMemcpyDeviceToHost) );
        			//fprintf(data->outlog, "MOD: MFDsedbud loop :%s\n", hipGetErrorString(hipGetLastError()));
	        		//printf("MOD: MFDsedbud loop :%s\n", hipGetErrorString(hipGetLastError()));

        			gridProgress += progressh2;

	        			if (progressh2 == 0)
	        					oneZero = 1;
	        					else {
	        							oneZero = 0;
	        							oneZero2 = 0;
	        					}

	        }  while ((progressh2 > 10 || !oneZero2)); // && ((double) *doneP * 100 / (gridRows * gridColumns)) < 99.0);

	        checkCudaErrors(hipMemcpy(progressd2, &progressh2, sizeof(progressh2), hipMemcpyHostToDevice) );

	        fprintf(data->outlog, "MOD:  loop :%s\n", hipGetErrorString(hipGetLastError()));

	        loop++;

} while (loop < loopMax || (loopForever && gridProgress) );

    checkCudaErrors(hipMemcpy(okGrid, ok, gridRows * gridColumns * sizeof(int),    hipMemcpyDeviceToHost) ) ;
    //checkCudaErrors(hipMemcpy(data->fa, device->fa, gridRows * gridColumns * sizeof(double), hipMemcpyDeviceToHost) ); //chnged from int !!!!!!!!!

  /* Free the GPU copies */
  hipFree(ok);
  hipFree(localOK_d);
  hipFree(progressd2);
  //free(&progressh);

  //printf("sum of gridProgress = %d\n", gridProgress);
  return gridProgress; // this does not seem to be copied back in this code??????
}


void sedmfdaccum(Data* data, Data* device)
{
  int ncols = data->mapInfo.width;
  int nrows = data->mapInfo.height;
  int fullsize = nrows * ncols;

  if (hipSuccess != hipSetDevice(CUDA_DEVICE))
  {
    printf("Unable to access CUDA card\n");
    return ;
  }

#ifndef PRODUCTION_RUN
  printf("\nGPU Card set for correctmfdflow\n\n");
#endif

  int x;
  float time;
  int percent;
  int gridprogress;
  int doneP;

  int *okgrid;
  int *localOK;

  hipEvent_t start, stop;

  percent = 0;
  gridprogress = 0;
  doneP = 0;

 // start the timer for correctflow;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  okgrid      = (int *)    malloc(fullsize * sizeof(int));
  localOK     = (int *)    malloc(fullsize * sizeof(int));

  if (okgrid == NULL || localOK == NULL ) {
    printf("Not enough memory to allocate grids 1\n");
    return ;
  }

  //set all host machine grids to zero
  for (x = 0; x < fullsize; ++x) {
    okgrid[x] = 0;
    localOK[x] = 0;
    if (data->mask[x] == 1) data->depoPtr[x] = 0.00001; //?????
  }

  hipMemcpy(device->depoPtr, data->depoPtr, fullsize * sizeof(double),   hipMemcpyHostToDevice);
#ifndef PRODUCTION_RUN
   printf("cuda error NOW1:%s\n", hipGetErrorString(hipGetLastError()));
#endif
   //  transport distance exponents
   //  double ddk = 0.95 * (23. / cell_size2); // travel distance parameter for diffuse flow -- normalized relative to 23 m plot
   //  double dck = 0.05 * (100. / cell_size2); // travel distance parameter for concentrated flow -- normalized relative to 100 m
   //  is the aim is to export ~5% of diffuse and 95% of concentrated after 100m?

    //probably not needed in this context??
  	if (data->ddk > 0.95)	data->ddk = 0.95;
  	if (data->dck > 0.9)	data->dck = 0.9;


  int loop = 0;

  int totalGP = 0;
  do {
    /* ---- beginning of first run ---- */
	//gridprogress = processWithPartitions(1, percent, 1568, 2704, GRDROWS, GRDCOLS, okgrid, fagrid, fdgrid, hvgrid, localOK, 169, 98, 1, BLOCKROWS, BLOCKCOLS, 1, &doneP);
	//printf(" arguments for gridprocess; Abr: %d, Abc: %d, GRDROWS: %d, GRDCOLS: %d, okgrid: %d, nrows: %d, ncols: %d \n",Abr, Abc, GRDROWS, GRDCOLS, okgrid, nrows, ncols);

	  //int processtheGrid(Data* data, Data* device, int loopMax, int percent, int gridRows, int gridColumns, int* okGrid,  int* localOK, int blockRows, int blockColumns, int dimBlock3, int* doneP)

	  gridprogress = processtheGrid(data, device, 1, percent, nrows, ncols, okgrid,  localOK, 16, 16, 1, &doneP, data->ddk, data->dck, data->dgk ) ;

	  /*printf("gridProgress = %d\n", gridprogress);*/
    /* ---- gap between runs ---- */
	//printf("cuda error NOW1:%s\n", hipGetErrorString(hipGetLastError()));
	//printf("half way there \n");

	if (gridprogress != 0)
	// gridprogress += processWithPartitions(1, percent,  672, 5408, GRDROWS, GRDCOLS, okgrid, fagrid, fdgrid, localOK, 338, 42, 1, BLOCKROWS, BLOCKCOLS, 1, &doneP);
	gridprogress += processtheGrid(data, device, 1, percent, nrows, ncols, okgrid,  localOK, 16, 16, 1, &doneP, data->ddk, data->dck, data->dgk) ;
   // printf("gridProgress = %d\n", gridprogress);
	//printf("cuda error NOW2:%s\n", hipGetErrorString(hipGetLastError()));
    totalGP += gridprogress;
    //printf("%d / %d (%f %%)\n", totalGP, nrows * ncols, (double) totalGP / (nrows * ncols) * 100);
    /* ---- end of second run ---- */
    loop ++;
  } while (gridprogress > 0); // && (double) totalGP / (GRDCOLS * GRDROWS)  < 0.99);


  hipMemcpy(data->depoPtr, device->depoPtr, fullsize * sizeof(double),   hipMemcpyDeviceToHost);
#ifndef PRODUCTION_RUN
  printf("cuda error NOW3:%s\n", hipGetErrorString(hipGetLastError()));
 // printf("Left to get = %d\n", nrows * ncols - totalGP);
  printf("About to finish sediment accumulation \n");
  //fflush(stdout);
#endif

  int count = 0;
  for (int r = 0; r < nrows; r++) {
	  for (int c = 0; c < ncols; c++) {
		  if (okgrid[r * ncols + c] != 1) {
			 // printf("Cell at [%d,%d] has not been computed!\n", r, c);
			  count ++;
		  }
	  }
  }
  fprintf(data->outlog, "Number of actual cells not computed = %d\n", count);


	thrust::device_ptr<double> deptot_d = thrust::device_pointer_cast(device->depoPtr);
	hipSetDevice(0);
	data->totD = thrust::reduce(deptot_d, deptot_d + fullsize, (double) 0);
	fprintf(data->outlog, "total Dep from thrust is %10.8lf \n", data->totD);

	if (data->totD == NAN)
	{
		printf("data from totD is NaN \n");
		exit(0);
	}

	  free(okgrid);
	  free(localOK);


  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  fprintf(data->outlog, "time to complete sediment accumulation algorithm %.6f s\n\n", time / 1000.0);

  return;
}
