
//#include "lem.h"
#include "erosion.h"
#include <math.h>
#include "device_constants.cuh"
#include "config.h"

#include "updates.h"
#include "mfd_accum.h"

void erosionGPU(Data* data, Data* device, int iter)
{
	int ncell_x = data->mapInfo.width;
	int ncell_y = data->mapInfo.height;
	int full_size = ncell_x * ncell_y;

	hipEvent_t start, stop;
	float time;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

   if (hipSuccess != hipSetDevice(CUDA_DEVICE)){
    printf("Unable to access CUDA card\n");
    exit(0);
  }

	size_t freenow, total; 
		
	fprintf(data->outlog, "MOD: Starting Model Process Routines \n");

	calc_diff_erosion(data, device);

	calc_conc_erosion(data, device);
	calc_gelifluction(data, device);

	fflush(data->outlog);

	//calc_sedflux(data, device);

	checkCudaErrors( hipMemcpy ( device->mask, data->mask, full_size * sizeof(int), hipMemcpyHostToDevice) );

	// sedmfdaccum(data, device);
	fprintf(data->outlog, "MOD: returned from sedmfdaccum :%s\n", hipGetErrorString(hipGetLastError()));

	checkCudaErrors( hipMemcpy ( data->eroPtr,   device->eroPtr,   full_size * sizeof(double), hipMemcpyDeviceToHost) );
	checkCudaErrors( hipMemcpy ( data->geliPtr,  device->geliPtr,  full_size * sizeof(double), hipMemcpyDeviceToHost) );
	checkCudaErrors( hipMemcpy ( data->inciPtr,  device->inciPtr,  full_size * sizeof(double), hipMemcpyDeviceToHost) );
	checkCudaErrors( hipMemcpy ( data->depoPtr,  device->depoPtr,  full_size * sizeof(double), hipMemcpyDeviceToHost) );
	checkCudaErrors( hipMemcpy ( data->SlopePtr, device->SlopePtr, full_size * sizeof(double), hipMemcpyDeviceToHost) );
	fprintf(data->outlog, "MOD: ero/inc/dep/slope memcopy :%s\n", hipGetErrorString(hipGetLastError()));

	calc_dz(data,device); // now includes gelifluction erosion

	checkCudaErrors( hipMemcpy ( data->dz, device->dz, full_size * sizeof(double), hipMemcpyDeviceToHost) );

	// Now add in weathering products and update cell calibre and cell moisture data

	calc_weathering(data, device);

	// now copy back all updated matrices
	checkCudaErrors( hipMemcpy ( data->finesPtr, device->finesPtr, full_size * sizeof(double), hipMemcpyDeviceToHost) );
	checkCudaErrors( hipMemcpy ( data->stonePtr, device->stonePtr, full_size * sizeof(double), hipMemcpyDeviceToHost) );
	checkCudaErrors( hipMemcpy ( data->soilTPtr, device->soilTPtr, full_size * sizeof(double), hipMemcpyDeviceToHost) );
	checkCudaErrors( hipMemcpy ( data->soilMPtr, device->soilMPtr, full_size * sizeof(double), hipMemcpyDeviceToHost) );
	checkCudaErrors( hipMemcpy ( data->weatherC, device->weatherC, full_size * sizeof(double), hipMemcpyDeviceToHost) );
	checkCudaErrors( hipMemcpy ( data->weatherP, device->weatherP, full_size * sizeof(double), hipMemcpyDeviceToHost) );
	fprintf(data->outlog, "MOD: fines/stone/soilT/soilM/weatherC and P memcopy back :%s\n", hipGetErrorString(hipGetLastError()));
	
	// Now update the surface height
	update_newSurface(data, device, iter);

	// Now update the nutrients on surface and in soil profile
	update_nutrients(data, device);

	checkCudaErrors( hipMemcpy ( data->soilBPtr, device->soilBPtr, full_size * sizeof(double), hipMemcpyDeviceToHost)) ;
	checkCudaErrors( hipMemcpy ( data->nutPtr,   device->nutPtr,   full_size * sizeof(double), hipMemcpyDeviceToHost) );
	fprintf(data->outlog, "MOD: conc_soilB/nutB copyback :%s\n", hipGetErrorString(hipGetLastError()));

	// Now grow the vegetation
	update_vegetation(data,device);

	checkCudaErrors( hipMemcpy( data->TotBPtr,  device->TotBPtr,  full_size * sizeof(double), hipMemcpyDeviceToHost) );
	fprintf(data->outlog, "MOD: mem copyback TotBn :%s\n", hipGetErrorString(hipGetLastError()));

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

#ifndef PRODUCTION_RUN
	printf("Time to complete model calculations %.6f s\n\n", time / 1000.0);
#endif

	fprintf(data->outlog, "MOD: time to complete flow accumulation %.6f s\n", time / 1000.0);

	hipMemGetInfo(&freenow, &total);
	fprintf(data->outlog, "MOD: Memory on CUDA card free at end of erosion: %zd total: %zd\n\n",freenow/1024,total/1024);

}


